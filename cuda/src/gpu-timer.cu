hipEvent_t tick, tock;
hipEventCreate(&tick);
hipEventCreate(&tock);

hipEventRecord(tick, 0);
kernel<<<griddim,blockdim>>> (foo, bar);
hipEventRecord(tock, 0);

hipEventSynchronize(tock);

float time;
hipEventElapsedTime(&time, tick, tock);

hipEventDestroy(tick);
hipEventDestroy(tock);
