hipEvent_t tick, tock;
hipEventCreate(&tick);
hipEventCreate(&tock);

hipEventRecord(tick, 0);
kernel<<<grid,threads>>> (arg1, arg2);
hipEventRecord(tock, 0);

hipEventSynchronize(tock);

float time;
hipEventElapsedTime(&time, tick, tock);

hipEventDestroy(tick);
hipEventDestroy(tock);
