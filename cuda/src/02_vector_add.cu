// Adapted from "CUDA by Example"


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vec_add(const int *a, const int *b, int *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

int main() {
  const int n = 16;

  int *a = (int *) malloc(n * sizeof(int));
  int *b = (int *) malloc(n * sizeof(int));
  int *c = (int *) malloc(n * sizeof(int));

  for (int i=0; i < n; ++i) {
    a[i] = -i;
    b[i] = i * i;
  }

  int *d_a, *d_b, *d_c;
  hipMalloc((void **) &d_a, n * sizeof(int));
  hipMalloc((void **) &d_b, n * sizeof(int));
  hipMalloc((void **) &d_c, n * sizeof(int));

  hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

  vec_add<<<1,n>>>(d_a, d_b, d_c, n);

  hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

  for (int i=0; i < n; ++i) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  free(a);
  free(b);
  free(c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
